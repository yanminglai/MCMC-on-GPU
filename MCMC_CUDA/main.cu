#include <iostream>
#include <fstream>
#include "metropolis_hastings.h"

using namespace std;

int main(int argc, char* argv[]) {
    const int dimension = 2;
    const int num_samples = argc>1 ? atoi(argv[1]) : 100;
    // Memory allocation
    float** samples;
    hipMallocManaged(&samples, num_samples*sizeof(float*));
    
    for(int i=0; i<num_samples; i++) {
        hipMallocManaged(&samples[i], dimension*sizeof(float));
    }

    metropolis_hastings<<<1,1>>>(num_samples, dimension, samples);
    hipDeviceSynchronize();

    ofstream output_file;
    output_file.open("samples.csv");
    for(int i=0; i<num_samples; i++) {
        for(int j=0; j<dimension-1; j++) {
            output_file<<samples[i][j]<<",";
        }
        output_file<<samples[i][dimension-1]<<"\n";
    }
    output_file.close();
    return 0;
}