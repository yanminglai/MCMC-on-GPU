#include "hip/hip_runtime.h"
#include "distribution_function.h"

__device__
float distribution_function(float x[]) {
    float exponent = -((x[0]-ux) * (x[0]-ux))/(2*sx*sx) - ((x[1]-uy) * (x[1]-uy))/(2*sy*sy);
    float probability = exp(exponent);
    return probability;
}